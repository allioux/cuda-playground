#include <iostream>
#include "utils.h"

void gpuMemInfos() {
    size_t* free;
    size_t* total;
    gpuErrchk(hipMallocManaged(&free, sizeof(size_t)));
    gpuErrchk(hipMallocManaged(&total, sizeof(size_t)));
    gpuErrchk(hipMemGetInfo(free, total));
    std::cout << "Free: " << *free << " | Total: " << *total << "\n";
}